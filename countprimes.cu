#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <limits.h>

#include <cutil_inline.h>
#include <countprimes_kernel.cu>
#include <defs.h>


void findPrimes(uint64 ulimit, byte* array) {
    CLR_BIT(array, 1-1);
    for(uint64 i = 2; i <= ulimit; i++)
        SET_BIT(array, i-1);
    uint64 thisFactor = 2;
    uint64 thisMark;
    const uint32 sqrt_ulimit = (uint32)floor(sqrt((double)ulimit));
    while(thisFactor <= sqrt_ulimit) {
        thisMark = thisFactor + thisFactor;
        while(thisMark <= ulimit) {
            CLR_BIT(array, thisMark-1);
            thisMark += thisFactor;
        }
        // Search for the next prime divisor
        do thisFactor++; while(GET_BIT(array, thisFactor-1) == 0);
    }
}

void markPrimesPattern(uint64 llimit, uint64 ulimit, uint64 lastFactor, byte* precomputed_primes, byte* array) {
    assert(llimit == 1);
    for(uint64 i = llimit; i <= ulimit; i++)
        SET_BYTE(array, i-llimit);
    uint64 thisFactor = 2;
    uint64 thisMark;
    const uint32 sqrt_ulimit = (uint32)floor(sqrt((double)ulimit));
    while(thisFactor <= lastFactor && thisFactor <= sqrt_ulimit) {
        thisMark = llimit - 1 + thisFactor;
        while(thisMark <= ulimit) {
            CLR_BYTE(array, thisMark-llimit);
            thisMark += thisFactor;
        }
        do  // Search for the next prime divisor in precomputed_primes
            thisFactor++;
        while(GET_BIT(precomputed_primes, thisFactor-1) == 0);
    }
}

void countPrimes_range(     const uint64 llimit,
                            const uint64 ulimit,
                            const byte* precomputed_primes,
                            const uint32 firstFactor,
                            const uint32 precomputed_top,
                            byte* array     )
{
//     printf("GOLDEN: [%llu, %llu] %u %u\n", llimit, ulimit, firstFactor, precomputed_top);
    assert(llimit >= 2);
    uint32 thisFactor = firstFactor;
    const uint32 sqrt_ulimit = (uint32)floor(sqrt((double)ulimit));
    while(thisFactor <= sqrt_ulimit) {
//         printf("GOLDEN: thisFactor=%u, precomputed_top=%u, sqrt_ulimit=%u\n", thisFactor, precomputed_top, sqrt_ulimit);
        uint64 thisMark = llimit;
        while(thisMark % thisFactor)
            thisMark++;
        while(thisMark <= ulimit) {
            CLR_BYTE(array, thisMark-llimit);
            thisMark += thisFactor;
        }
        do  // Search for the next prime divisor in precomputed_primes
            thisFactor++;
        while(GET_BIT(precomputed_primes, thisFactor-1) == 0);
        assert(thisFactor <= precomputed_top);
    }
}


int main(int argc, char *argv[]) {
    const uint16 num_mp = 64;
    const uint16 num_threads = 64;
    const uint32 lastFactor_pre = 13;
    const uint32 firstFactor_sieve = 17;
    // LCM(2,3,5,...initial primes, 8, 64) (to align with 64-byte boundary of device memory)
    const uint32 num_bytes_pattern = 3*5*7*11*13*64;

    double ll_double, ul_double;   // upper and lower limits, both inclusive
    assert(argc >= 3);
    sscanf(argv[1], "%lf", &ll_double);
    sscanf(argv[2], "%lf", &ul_double);
    const uint64 llimit = (ll_double < 2.0) ? (uint64)2 : (uint64)ll_double;
    const uint64 ulimit = (uint64)ul_double;
    const uint32 sqrt_ulimit = (uint32)floor(sqrt((double)ulimit));
    assert(llimit <= ulimit);
    assert(ulimit - llimit >= (uint64)1.9e5);
    assert(ulimit <= (CONSTANT_MEM_SIZE*8)*(CONSTANT_MEM_SIZE*8)); // 2^38 = 274877906944
    printf("Counting primes in the interval [%llu, %llu]...\n", llimit, ulimit);

// precomputing primes upto sqrt(ulimit)
    uint32 precomputed_top = (uint32)(ceil(floor(sqrt((double)ulimit))/(num_mp*8.0))*(num_mp*8));
    uint32 num_bytes_pre = (uint32)ceil(precomputed_top/8.0);   // 8 numbers per byte
    printf("num_bytes_pre = %u Bytes, %.2fKB\n", num_bytes_pre, num_bytes_pre/1024.0);
    byte* precomputed_primes = NULL;
    precomputed_primes = (byte*)malloc(num_bytes_pre);          // bit-wise array
    assert(precomputed_primes != NULL);
    uint32 timer_pre = 0;
    cutilCheckError(cutCreateTimer(&timer_pre));
    cutilCheckError(cutStartTimer(timer_pre));
    findPrimes(precomputed_top, precomputed_primes);    // call the function
    cutilCheckError(cutStopTimer(timer_pre));

// counting some primes from the precomputed primes list
    uint64 prime_precounter = 0;
    if(llimit <= precomputed_top) {
        for(uint64 i = llimit; i <= precomputed_top; i++)
            if(GET_BIT(precomputed_primes, i-1))
                prime_precounter++;
        printf("  %llu primes found from the precomputed list [%llu, %u]\n", prime_precounter, llimit, precomputed_top);
    }

// precomputing pattern of non-primes which are multiples of some intial primes
    byte* precomputed_pattern = NULL;           // byte-wise array
    precomputed_pattern = (byte*)malloc(num_bytes_pattern);
    assert(precomputed_pattern != NULL);
    cutilCheckError(cutStartTimer(timer_pre));
    markPrimesPattern(1, num_bytes_pattern, lastFactor_pre, precomputed_primes, precomputed_pattern);   // call the function
    cutilCheckError(cutStopTimer(timer_pre));
    float time_pre = cutGetTimerValue(timer_pre);
    printf("CPU: %fms and %u bytes taken to precompute primes between [1, %u] and marking the pattern [1, %u] \n", time_pre, num_bytes_pre, precomputed_top, num_bytes_pattern);

// CPU memory allocation and filling it with precomputed_pattern
    uint64 num_bytes = ulimit-llimit+1;
    printf("num_bytes = %llu, %.2fMB\n", num_bytes, num_bytes/1024.0/1024.0);
    byte* all_primes = NULL;
    all_primes = (byte*)malloc(num_bytes);
    assert(all_primes != NULL);
    int patternboundray_2_llimit = ((llimit-1) % num_bytes_pattern);
    int llimit_2_patternboundray = num_bytes_pattern - patternboundray_2_llimit;
    uint32 start_address = 0;
    if(patternboundray_2_llimit != 0) {
        memcpy(all_primes, precomputed_pattern+patternboundray_2_llimit, llimit_2_patternboundray);
        start_address = llimit_2_patternboundray;
    }
    for(uint32 i = start_address; i < num_bytes; i += num_bytes_pattern) {
        if(i+num_bytes_pattern > num_bytes)
            memcpy(all_primes+i, precomputed_pattern, num_bytes-i);
        else
            memcpy(all_primes+i, precomputed_pattern, num_bytes_pattern);
    }

// now using GPU...
    // copy precomputed_primes in device constant memory
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_precomputed_primes), precomputed_primes, num_bytes_pre));
    byte* d_all_primes = NULL;
    cutilSafeCall(hipMalloc(&d_all_primes, num_bytes));
    assert(d_all_primes != NULL);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_all_primes, all_primes, num_bytes, hipMemcpyHostToDevice));
    uint32 timer_gpu = 0; cutilCheckError(cutCreateTimer(&timer_gpu));
    // cook the kernel
    printf("launching kernel with %u blocks and %u threads...\n", num_mp, num_threads);
    primeKernel<<<num_mp, num_threads>>>(llimit, ulimit, d_all_primes, firstFactor_sieve, num_bytes_pre, sqrt_ulimit, 1);
    cutilCheckError(cutStartTimer(timer_gpu));
    // launch the kernel
    {
        primeKernel<<<num_mp, num_threads>>>(llimit, ulimit, d_all_primes, firstFactor_sieve, num_bytes_pre, sqrt_ulimit, 0);
        cutilCheckMsg("Kernel execution failed");
        hipDeviceSynchronize();
    }
    cutilCheckError(cutStopTimer(timer_gpu));
    float time_gpu = cutGetTimerValue(timer_gpu);

    byte* all_primes_gpu_result = NULL;
    all_primes_gpu_result = (byte*)malloc(num_bytes);
    assert(all_primes_gpu_result != NULL);
    cutilSafeCall(hipMemcpy(all_primes_gpu_result, d_all_primes, num_bytes, hipMemcpyDeviceToHost));
    // counting primes
    uint64 prime_counter_gpu;
    prime_counter_gpu = prime_precounter;
    for(uint64 i = llimit; i <= ulimit; i++)
        if(GET_BYTE(all_primes_gpu_result, i-llimit))
            prime_counter_gpu++;
    printf("GPU: %llu primes found between [%llu, %llu] in %.3f ms\n", prime_counter_gpu, llimit, ulimit, time_gpu);
    free(all_primes_gpu_result);

#if 1
// reference solution by CPU
    uint32 timer_cpu = 0;
    cutilCheckError(cutCreateTimer(&timer_cpu));
    cutilCheckError(cutStartTimer(timer_cpu));
    {
        countPrimes_range(llimit, ulimit, precomputed_primes, firstFactor_sieve, precomputed_top, all_primes);
    }
    cutilCheckError(cutStopTimer(timer_cpu));
    float time_cpu = cutGetTimerValue(timer_cpu);
    // counting primes
    uint64 prime_counter = prime_precounter;
    for(uint64 i = llimit; i <= ulimit; i++)
        if(GET_BYTE(all_primes, i-llimit))
            prime_counter++;
    printf("CPU: %llu primes found between [%llu, %llu] in %.3f ms\n", prime_counter, llimit, ulimit, time_cpu);
    if(prime_counter - prime_counter_gpu != 0)
        printf("ERROR: CPU crossed out %llu less non-primes than GPU\n", prime_counter - prime_counter_gpu);
    free(all_primes);
#endif

    return 0;
}
