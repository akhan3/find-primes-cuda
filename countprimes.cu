#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <math.h>

#include <cutil_inline.h>
#include <countprimes_kernel.cu>


#define SET_BIT(array, i) (array)[(i)>>3] |= 1<<((i)&0x07)
#define CLR_BIT(array, i) (array)[(i)>>3] &= ~(1<<((i)&0x07))
#define GET_BIT(array, i) ((array)[(i)>>3] & 1<<((i)&0x07)) >> ((i)&0x07)

typedef  unsigned long long uint64;
typedef  unsigned char byte;

void findPrimes (uint64 ulimit, byte* array) {
    CLR_BIT(array, 1-1);
    for(uint64 i = 2; i <= ulimit; i++)
        SET_BIT(array, i-1);
    uint64 thisFactor = 2;
    uint64 mark;
    while(thisFactor * thisFactor <= ulimit) {
        mark = thisFactor + thisFactor;
        while(mark <= ulimit) {
            CLR_BIT(array, mark-1);
            mark += thisFactor;
        }
        // Search for the next prime divisor
        do thisFactor++; while(GET_BIT(array, thisFactor-1) == 0);
        assert(thisFactor <= ulimit);
    }
}

void findPrimes_range (uint64 llimit, uint64 ulimit, byte* precomputed_primes, byte* array) {
    for(uint64 i = llimit; i <= ulimit; i++)
        SET_BIT(array, i-llimit);
    uint64 thisFactor = 2;
    uint64 mark;
    while(thisFactor * thisFactor <= ulimit) {
        mark = thisFactor;
        printf("thisFactor = %llu\n", thisFactor);
        while(mark <= ulimit) {
//             printf("thisFactor=%llu, mark=%llu, mark-1=%llu, (mark-1)>>3=%llu\n", thisFactor, mark, mark-1, (mark-1)>>3);
            CLR_BIT(array, mark-1);
            mark += thisFactor;
        }
        // Search for the next prime divisor in precomputed_primes
        do thisFactor++; while(GET_BIT(precomputed_primes, thisFactor-1) == 0);
        assert(thisFactor <= ulimit);
    }
}

void markPrimesPattern(uint64 llimit, uint64 ulimit, uint64 top_divisor, byte* precomputed_primes, byte* array) {
    for(uint64 i = llimit; i <= ulimit; i++)
        SET_BIT(array, i-llimit);
    uint64 thisFactor = 2;
    uint64 mark;
    while(thisFactor <= top_divisor && thisFactor * thisFactor <= ulimit) {
        mark = llimit - 1 + thisFactor;
        printf("thisFactor = %llu\n", thisFactor);
        while(mark <= ulimit) {
            printf("  thisFactor=%llu, mark=%llu, (mark-llimit)=%llu, (mark-llimit)>>3=%llu\n", thisFactor, mark, mark-llimit, (mark-llimit)>>3);
            CLR_BIT(array, mark-llimit);
            mark += thisFactor;
        }
        do  // Search for the next prime divisor in precomputed_primes
            thisFactor++;
        while(GET_BIT(precomputed_primes, thisFactor-1) == 0);
        assert(thisFactor <= ulimit);
    }
}


int main(int argc, char *argv[]) {
    uint64 llimit, ulimit;  // upper and lower limits, both inclusive
    float ll_float, ul_float;
    assert(argc == 3);
    sscanf(argv[1], "%f", &ll_float);
    sscanf(argv[2], "%f", &ul_float);
    llimit = (uint64)ll_float;
    ulimit = (uint64)ul_float;
    assert(llimit <= ulimit);
    assert(ulimit <= 274877906944);
    printf("Counting primes in the interval [%llu, %llu]...\n", llimit, ulimit);

    const uint64 SIXTYFOUR_KB = 65536;
    uint64 precomputed_top = SIXTYFOUR_KB * 8;    // 524,288
    byte precomputed_primes[SIXTYFOUR_KB];
    findPrimes(precomputed_top, &precomputed_primes[0]);

    uint64 prime_counter = 0;

//     if(ulimit <= precomputed_top) {
//         printf("No need to use GPU...\n");
//         prime_counter = 0;
//         for(uint64 i = llimit; i <= ulimit; i++)
//             if(GET_BIT(precomputed_primes, i-1))
//                 prime_counter++;
//         printf("%llu primes found between [%llu, %llu]\n", prime_counter, llimit, ulimit);
//         return 0;
//     }
//
//     if(llimit <= precomputed_top) {
//         printf("counting some primes from the precomputed list...\n");
//         prime_counter = 0;
//         for(uint64 i = llimit; i <= precomputed_top; i++)
//             if(GET_BIT(precomputed_primes, i-1))
//                 prime_counter++;
//         printf("%llu primes found between [%llu, %llu]\n", prime_counter, llimit, precomputed_top);
//     }


    byte* precomputed_pattern = 0;    // pattern of marked non-primes which are multiple of (2,3,5,7,11,13)
    precomputed_pattern = (byte*)malloc(15015);    // pattern of marked non-primes which are multiple of (2,3,5,7,11,13)
    llimit = 7;
    ulimit = 78;
    markPrimesPattern(llimit, ulimit, 3, &precomputed_primes[0], &precomputed_pattern[0]);

    prime_counter = 0;
    for(uint64 i = llimit; i <= ulimit; i++) {
        printf("%u", GET_BIT(precomputed_pattern, i-1));
        if(GET_BIT(precomputed_pattern, i-1))
            prime_counter++;
    }
    printf("\n%llu primes found between [%llu, %llu]\n", prime_counter, llimit, ulimit);


    // now using GPU...
//     uint64 ll_gpu = precomputed_top+1;
//     uint64 ul_gpu = ulimit;
//     cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_bitarray), precomputed_primes, SIXTYFOUR_KB));



    return 0;
}
